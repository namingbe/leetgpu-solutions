// CUDA 12.8.0
#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < height * width * 4 && x % 4 != 3) {
        image[x] = 255 - image[x];
    }
}
// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(unsigned char* image, int width, int height) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height * 4 + threadsPerBlock - 1) / threadsPerBlock;

    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(image, width, height);
    hipDeviceSynchronize();
}
